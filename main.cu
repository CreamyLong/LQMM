#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <time.h>
#include <hip/hip_runtime.h> //make_float3

#include "init.h"
#include "utils.h"
#include "move.h"
#include "force.h"
#include "macro.h"

// main function
int main(int argc, char** argv)
{
	/* phase seperation parameters for video:
	unsigned int nsteps = 200000;
	unsigned int nprint = 5000;
	float3 epsilon = make_float3(1.0, 0.2, 1.0);
	*/
	//running parameters

	/*
	unsigned�����þ��ǽ����������޷��Ż��� ���� int �͵ķ�Χ��-2^31 ~ 2^31 - 1����unsigned int�ķ�Χ��0 ~ 2^32��
	�޷��Ű汾���з��Ű汾����������޷��������ܱ���2�����з������͵����������ݡ�
	*/
	unsigned int np = 2700;                  // the number of particles
	unsigned int nsteps = 500;             // the number of time steps
	float dt = 0.001;                       // integration time step
	float rcut = 3.0;                       // the cutoff radius of interactions

	unsigned int nprint = 100;              //  period for data output
	unsigned int block_size = 128;          //  the number of threads in a block           

	clock_t start, end; // start time , end time

	
	
	float3 box = make_float3(15.0, 15.0, 15.0);     // box size in x, y, and z directions
	
	//Lennard-Jones����-��˹�������ܼ��㣬������ģ�����������Եķ��ӻ�ԭ�Ӽ��໥�������ܵ�һ���Ƚϼ򵥵���ѧģ�͡�
	/*
	    epsilon��������������
		epsilon.x for type 1.0 and 1.0;
		epsilon.y for type 1.0 and 2.0;
		epsilon.z for type 1.0 and 2.0
	*/
	float3 epsilon = make_float3(1.0, 0.5, 1.0);   
	
	/*
	    sigma�ǻ������õ���������Ϊ��ʱ���������
		sigma.x for type 1.0 and 1.0;
		sigma.y for type 1.0 and 2.0;
		sigma.z for type 1.0 and 2.0
	*/
	float3 sigma = make_float3(1.0, 1.0, 1.0);     // 
	
	float min_dis = sigma.x * 0.9;	// the minimum distance between particles for system generation
	
	float3 lj1, lj2;

	//��һ�����Ϊ�Ƕ�Ӧ�������ڽ�����ʱ�Ի����ų�Ϊ��������
	lj1.x = 4.0 * epsilon.x * pow(sigma.x, float(12.0));
	lj1.y = 4.0 * epsilon.y * pow(sigma.y, float(12.0));
	lj1.z = 4.0 * epsilon.z * pow(sigma.z, float(12.0));

	//�ڶ����Ӧ������Զ�����Ի�������������ͨ�������߶�˹����Ϊ��������
	lj2.x = 4.0 * epsilon.x * pow(sigma.x, float(6.0));
	lj2.y = 4.0 * epsilon.y * pow(sigma.y, float(6.0));
	lj2.z = 4.0 * epsilon.z * pow(sigma.z, float(6.0));

	//host memory allocation
	float4* h_r = (float4*)malloc(np * sizeof(float4));  // rx, ry, rz, type(0, 1, 2 ...)
	float4* h_v = (float4*)malloc(np * sizeof(float4));  // vx, vy, vz, mass
	float4* h_f = (float4*)malloc(np * sizeof(float4));  // fx, fy, fz, potential
	float* h_info = (float*)malloc(16 * sizeof(float));  // temperature, potential, energy ...


	//device memory allocation
	float4* d_r;
	float4* d_v;
	float4* d_f;
	float* d_info;
	float2* d_scratch;
	hipMalloc((void**)&d_r, np * sizeof(float4));     // rx, ry, rz, type(0, 1, 2 ...)
	hipMalloc((void**)&d_v, np * sizeof(float4));     // vx, vy, vz, mass
	hipMalloc((void**)&d_f, np * sizeof(float4));     // fx, fy, fz, potential
	hipMalloc((void**)&d_info, 16 * sizeof(float));	  // temperature, potential, energy ...
	hipMalloc((void**)&d_scratch, (np / block_size + 1) * sizeof(float2));	  // temporary data ...

	FILE* traj = fopen("traj.xyz", "w");                 // trajectory file in XYZ format that can be open by VMD

		/* generate system information */

	printf("Starting simulation with %d atoms for %d steps.\n", np, nsteps);
	printf("Generating system.\n", np, nsteps);


	init(np, h_r, h_v, box, min_dis);

	hipMemcpy(d_r, h_r, np * sizeof(float4), hipMemcpyHostToDevice);
	hipMemcpy(d_v, h_v, np * sizeof(float4), hipMemcpyHostToDevice);

	start = clock();

	/* main MD loop */
	printf("Running simulation.\n", np, nsteps);

	for (unsigned int step = 0; step <= nsteps; step++) //running simulation loop
	{
		/* first integration for velverlet */
		first_integration(np, dt, box, d_r, d_v, d_f, block_size);

		/* force calculation */
		force_calculation(np, box, lj1, lj2, d_r, d_f, rcut * rcut, block_size);

		/* compute temperature and potential */
		compute_info(np, d_v, d_f, d_scratch, d_info, block_size);

		/* second integration for velverlet */
		second_integration(np, dt, d_v, d_f, block_size);

		/* write output frames and system information, if requested */
		if ((step % nprint) == 0)
		{
			hipMemcpy(h_r, d_r, np * sizeof(float4), hipMemcpyDeviceToHost);
			hipMemcpy(h_info, d_info, 16 * sizeof(float), hipMemcpyDeviceToHost);
			output(traj, step, h_info, h_r, np);
			printf("time step %d \n", step);
		}
	}

	end = clock();
	double  duration = (double)(end - start) / CLOCKS_PER_SEC;

	printf("%f seconds\n", duration);

	fclose(traj);
	free(h_r);
	free(h_v);
	free(h_f);
	free(h_info);

	hipFree(d_r);
	hipFree(d_v);
	hipFree(d_f);
	hipFree(d_info);
	return 0;
}
